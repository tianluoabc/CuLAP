/*
 * Created by Ketan Date
 */

#include <iostream>
#include <fstream>
#include <ctime>
#include <sstream>
#include <omp.h>
#include "../include/structures.h"
#include "../include/variables.h"
#include "../include/helper_utils.h"
#include "../include/culap.h"
#include "../include/sfmt.h"

void modifyCosts(double *, int, int, int);

int main(int argc, char **argv)
{

	int size = atoi(argv[1]);
	int costrange = atoi(argv[2]);
	int problemcount = atoi(argv[3]);
	int repetitions = atoi(argv[4]);

	int numdev = atoi(argv[5]);
	int spcount = atoi(argv[6]);

	const char *filename = argv[7];

	int multiplier = 1;

	int init_assignments = 0;
	int stepcounts[7];
	double steptimes[9];

	std::fill(stepcounts, stepcounts + 7, 0);

	std::stringstream logpath;
	int problemsize = size;

	costrange = problemsize * 10;

	int *row_assignments;
	double *row_duals, *col_duals;
	double *cost_matrix;
	double *obj_val;

	int devid = 0;

	hipSetDevice(devid);

	double *h_cost = new double[spcount * size * size];
	int *h_ass = new int[spcount * size];
	double *h_row_dual = new double[spcount * size];
	double *h_col_dual = new double[spcount * size];

	hipMalloc((void **)&row_assignments, spcount * size * sizeof(int));
	hipMalloc((void **)&row_duals, spcount * size * sizeof(double));
	hipMalloc((void **)&col_duals, spcount * size * sizeof(double));
	hipMalloc((void **)&cost_matrix, spcount * size * size * sizeof(double));
	hipMalloc((void **)&obj_val, spcount * sizeof(double));

	hipMemset(row_assignments, -1, spcount * size * sizeof(int));
	hipMemset(row_duals, 0, spcount * size * sizeof(double));
	hipMemset(col_duals, 0, spcount * size * sizeof(double));
	hipMemset(obj_val, 0, spcount * sizeof(double));

	//	readFile(h_cost, filename, spcount);
	generateProblem(h_cost, spcount, size, costrange);

	hipMemcpy(cost_matrix, h_cost, spcount * size * size * sizeof(double), hipMemcpyHostToDevice);

	for (int i = 0; i < repetitions; i++)
	{

		std::cout << "Size: " << problemsize << "\tCostrange: [0, " << costrange << "]" << std::endl;

		double start = omp_get_wtime();

		size_t total, free1, free2;

		//		hipMemGetInfo(&free1, &total);

		CuLAP lpx(problemsize, spcount, devid, false, stepcounts);
		lpx.solve(cost_matrix, row_assignments, row_duals, col_duals, obj_val);

		//		hipMemGetInfo(&free2, &total);

		//		std::cout << "Leakage: " << free1 - free2 << "B" << std::endl;

		double end = omp_get_wtime();

		double total_time = (end - start);

		printDebugArray(obj_val, spcount, "obj_val");

		std::cout << "Itn count: " << stepcounts[3] << "\tOriginal time: " << total_time << " s" << std::endl;

		/*		hipMemcpy(h_ass, row_assignments, spcount * size * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(h_row_dual, row_duals, spcount * size * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(h_col_dual, col_duals, spcount * size * sizeof(double), hipMemcpyDeviceToHost);

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////

		modifyCosts(h_cost, spcount, size, size / 10);
		hipMemcpy(cost_matrix, h_cost, spcount * size * size * sizeof(double), hipMemcpyHostToDevice);

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////

		hipMemcpy(row_assignments, h_ass, spcount * size * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(row_duals, h_row_dual, spcount * size * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(col_duals, h_col_dual, spcount * size * sizeof(double), hipMemcpyHostToDevice);

		start = omp_get_wtime();

		CuLAP lpy(problemsize, spcount, devid, false);
		lpy.solve(cost_matrix, row_assignments, row_duals, col_duals, obj_val);

		end = omp_get_wtime();

		printDebugArray(obj_val, spcount, "obj_val");
//		printDebugArray(row_assignments, spcount * size, "assignment");

		total_time = (end - start);

		std::cout << "Re-solve time: " << total_time << " s" << std::endl;

		///////////////////////////////////////////////////////////////////////////////////////////////////////////////

		hipMemcpy(row_assignments, h_ass, spcount * size * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(row_duals, h_row_dual, spcount * size * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(col_duals, h_col_dual, spcount * size * sizeof(double), hipMemcpyHostToDevice);

		start = omp_get_wtime();

		CuLAP lpz(problemsize, spcount, devid, true);
		lpz.solve(cost_matrix, row_assignments, row_duals, col_duals, obj_val);

		end = omp_get_wtime();

		total_time = (end - start);

		printDebugArray(obj_val, spcount, "obj_val");
//		printDebugArray(row_assignments, spcount * size, "assignment");

		std::cout << "Dynamic time: " << total_time << " s" << std::endl;
*/
	}

	hipFree(row_assignments);
	hipFree(row_duals);
	hipFree(col_duals);
	hipFree(cost_matrix);
	hipFree(obj_val);

	delete[] h_cost;
	delete[] h_ass;
	delete[] h_row_dual;
	delete[] h_col_dual;

	return 0;
}

void modifyCosts(double *cost_matrix, int SP, int N, int mod_count)
{

	CRandomSFMT randomGenerator(SEED);

	for (int i = 0; i < SP; i++)
	{

		for (int j = 0; j < mod_count; j++)
		{

			//double val = randomGenerator.Random();
			double sign = randomGenerator.Random();
			double val = (double)randomGenerator.IRandomX(0, 20);
			double delta = (sign < 0.5) ? -val : val;
			long id = (long)randomGenerator.IRandomX(0, N * N);
			long tid = i * N * N + id;

			cost_matrix[tid] += delta;
		}
	}
}
